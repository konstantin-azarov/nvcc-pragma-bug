#include "hip/hip_runtime.h"
#include <iostream>
#include <stdio.h>

// Uncomment the pragma below to break things 
#define EIGEN_DISABLE_HD_WARNING // #pragma hd_warning_disable
#include <Eigen/Eigen>

namespace e = Eigen;

__global__ void Transform(e::Affine3f pose, e::Vector3f v) {
  e::Vector3f res = pose * v;

  printf("Result: (%f, %f, %f)\n", v.x(), v.y(), v.z());
}

int main() {
  e::Affine3f pose = e::Affine3f::Identity();

  Transform<<<1, 1>>>(pose, e::Vector3f(1, 0, 0));
  hipDeviceSynchronize();

  return 0;
}
